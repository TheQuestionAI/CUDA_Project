#include "hip/hip_runtime.h"
/*
Multiple GPUs-based CUDA implementation of 2D acoustic wave propagation using finite-difference scheme in time domain. 

The code is solving second order 2D wave equation:
	d^2u/dx^2 + d^u^2/du^2 = v^(-2) * d^2u/dt^2

	u = u(x,y; t) 		=> the wave field
	v = v(x,y) 			=> the constant wave velocity in medium.

Finite Difference:
	We use 17-point stencil template to approximate the partial derivative at a single wave field point.
						*
						*
						*
						*
				* * * * + * * * * 
						*
						*
						*
						*

Multiple GPUs implementation, each will be responsible for one sub-wave-field domain.

                  GPU 0                                          GPU 1 									    ... 				GPU N
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | | 		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |
      | | | | * * * * * * * * * * * - - - - | | | |     | | | | - - - - * * * * * * * * * * * - - - - | | | |		...  	| | | | - - - - * * * * * * * * * * * | | | |

      padding		body     		halo    padding     padding   halo      	body 		   halo   padding 				padding   halo			body 		  padding

*/


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// padding for fd scheme.
#define PAD 4
#define PAD2 8

// define wave equation and fd coefficients
#define a0  -2.8472222f
#define a1   1.6000000f
#define a2  -0.2000000f
#define a3   0.0253968f
#define a4  -0.0017857f

#define v 0.12f 		// wave velocity square.

// define thread block dimension, padding
#define BDIMX 256

// store the coefficient and wave evelocity to constant memory
__constant__ float dc_coeff[5];
__constant__ float dc_v;


// Check error codes for CUDA functions
void CUDA_ERROR_CHECK(hipError_t err) 
{
    if (err != hipSuccess) 
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}


// setup constant variables.
void setup_constant_coefficient()
{
	const float h_coef[5] = {a0, a1, a2, a3, a5};
	const float h_v = v;
	CUDA_ERROR_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dc_coef), h_coef, 5 * sizeof(float)) );
    CUDA_ERROR_HANDLE( hipMemcpyToSymbol(HIP_SYMBOL(dc_v), h_v, sizeof(float)) );
}

// calculate each intervals for the halo region and body region.
inline void calculate_halo_body_interval(int* halo_start, int* halo_end, int* body_start, int* body_end, const int ngpus, const int iny) 
{
	if(ngpus == 0)		// one gpu special case
	{
        body_start[idx] = PAD;
        body_end[idx]   = iny - PAD2 - 1;

        halo_start[idx] = iny - PAD2;
        halo_end[idx]   = iny - PAD - 1;

        return;		
	}

    // halo regions
    for(int idx = 0; idx < 2*(ngpus-1); ++idx) 
    {
        if (idx == 0)				// GPU 0 -> only right hand side has halo region 
        {
            body_start[idx] = PAD;
            body_end[idx]   = iny - PAD2 - 1;

            halo_start[idx] = iny - PAD2;
            halo_end[idx]   = iny - PAD - 1;

        }
        else if(idx == ngpus - 1)		// GPU N -> only left hand side has halo region
        {
            halo_start[idx] = PAD;      
            halo_end[idx]   = PAD2 - 1;

            body_start[idx] = PAD2;
            body_end[idx]   = iny - PAD - 1;  
        }
        else  							// GPU 1 ... N-1 -> both left and right side have halo region
        {	// left halo
            halo_start[idx] = PAD;      
            halo_end[idx]   = PAD2 - 1;
            // body
            body_start[idx] = PAD2;
            body_end[idx]   = iny - PAD2 - 1;          	
            // right halo
            halo_start[++idx] = iny - PAD2;
            halo_end[++idx]   = iny - PAD - 1;

        }

}

// re-visited
inline void calcSkips(int* src_skip, int* dst_skip, const int nx, const int iny) 
{
    src_skip[0] = nx * (iny - NPAD2);     // 计算源GPU内点区域所有点数. iny - NPAD2即内点区域y轴区间的长度. 记住只有俩个GPU, 也即任意一GPU的计算只有一边有halo区域. 
    dst_skip[0] = 0;                      // 目的GPU什么都不跳过.
    src_skip[1] = NPAD * nx;              // 计算padding/halo区域所有点数
    dst_skip[1] = (iny - NPAD) * nx;      // 计算目的GPU 内点区域 + halo区域所有点数. iny - NPAD即内点区域+halo区域的y轴长度.
}

// intial wavelet for the wave field at time = 0. Need to select the central place GPU to set.
__global__ void kernel_add_initial_wavelet(float* d_u, float init_wavelet, const int nx, const int iny, const int ngpus) 
{
    int src_ypos = (ngpus % 2 == 0 ? iny : iny / 2);			// ngpus even or odd, the central for y is differnt.
    int src_xpos = nx / 2;
    
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;	// Align with the thread block and thread grid dimension selected.
    unsigned int idx = src_ypos * nx + ix;		// 1D global index for wave field array
    if(ix == src_xpos) 							// put the initial wave at the center of wave field u(x,y; t)
		d_u[idx] += init_wavelet;
}

/*
    const int iny    = ny / ngpus + NPAD * 2;                           // 简化问题, 统一以y轴划分每个分区, 在边界左右两边都有NPAD的padding点. iny表示的是每个GPU分配的分区的y维度网格大小, 包括padding区域.
    size_t isize     = nx * iny;                                        // 每个分区需要的网格点总数量.
    size_t ibyte     = isize * sizeof(float);                           // 每个分区需要的设备内存字节数.
    size_t iexchange = NPAD * nx * sizeof(float);                       // 需要在分区间进行数据交换的切片区域占据的内存字节数.
    hipMalloc((void**)&d_u1[i], ibyte);    g_u1 = &d_u1[i];
    hipMalloc((void**)&d_u2[i], ibyte)     g_u2 = &d_u2[i];
*/
// The core finite difference kernel function. 
__global__ void kernel_2dfd(float *d_u1, float *d_u2, const int nx, const int iStart, const int iEnd) {

    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;           // 计算当前线程在线程网格的1D线性化全局x轴索引. 注意线程网格使用的x-y轴 和 波场网格使用的x-y轴 刚好相反.

    __shared__ float line[BDIMX + PAD2];                               // 共享内存的大小基本上是跟着thread block的大小走的. 这里采用的是1D thread block, 因此共享内存简单起见也会被定义成1D的.
    // 注意在line两边加padding, 这样的话就不用考虑边界case了. 所有点都用同样的代码模式.

    // smem idx for current point
    unsigned int stx = threadIdx.x + PAD;                              // 计算当前线程对应的共享内存索引. offset NPAD=4必加, 最开头NPAD个点是padding.
    unsigned int idx = ix + iStart * nx;                                // idx计算的是当前线程对应的GPU细分区域下的全局波场数组1D线性化索引. 特别注意, wave field数组是一个1Darray, 即使wave field是2D的.
    // iStart * nx 即是y轴下index = iStart之前的波场2D区域的点数(当前GPU的细分区域下). ix表示的是当前线程网格下的线程的1D线性化索引. 
    // 两者相加得到idx, idx即是当前线程要计算的对应的GPU细分区域下的2D全局波场点的1D线性化索引.

    // register for y value. 这里直接使用寄存器, 本质是与共享内存一样, 把全局内存的访问拉近到对寄存器的访问.
    float yval[9];      // y轴方向使用寄存器存储, 每个线程计算一个wave field点, 而y轴方向需要9点求偏导数.
    #pragma unroll
    for (unsigned int i = 0; i < 8; ++i) 
        yval[i] = d_u2[idx + (i - 4) * nx];   // 这里并没有对全局内存的连续访问啊! 这里只写入了y轴上的8个点 => + + + + * + + + => 第9个点放置在了for loop里写入.

    // skip for the bottom most y value
    int iskip = PAD * nx;  // skip掉当前y轴4个点形成的2D区域, 这样我们可以到第九个点进行写入寄存器.

    #pragma unroll 9
    for (unsigned int iy = iStart; iy < iEnd; ++iy) {  // 循环使用共享内存, 以节约空间. 这里一个线程一次性将计算给定2D波长区域的一整条y轴区间长度的点.
        // get yval[8] here
        yval[8] = d_u2[idx + iskip];

        // read halo part
        if(threadIdx.x < PAD) {      // 索引小于NPAD就是halo区域. 这里是对共享内存区域的写入.
            line[threadIdx.x]  = d_u2[idx - PAD];
            line[stx + BDIMX]  = d_u2[idx + BDIMX];   // halo区域的的点将会多负责两个slot的共享内存写入.
        }

        line[stx] = yval[4];
        __syncthreads();              // 每个线程写入共享内存一个slot, 最后需要同步以使得线程对共享内存的写入全部完成.

        // 8rd fd operator. 这里开始真正的有限差分计算求当前时刻波场状态了.
        if ( (ix >= PAD) && (ix < nx - PAD) ) {     // 特别注意, 只有 内点区域 + halo区域要计算波场.
            // center point
            float tmp = coef[0] * line[stx] * 2.0f;   // 中心点会被用2次. 一次x偏导, 一次y偏导.
            #pragma unroll
            for(unsigned int d = 1; d <= 4; ++d) {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);			// d^2u/dx^2
             	tmp += coef[d] * (yval[4 - d] + yval[4 + d]);				// d^u/dy^2
            }

            // time dimension
            d_u1[idx] = 2.0f * yval[4] - g_u1[idx] + dc_v * tmp;    // 有限差分计算相加.
        }

        #pragma unroll 8
        for (int i = 0; i < 8 ; i++) {
            yval[i] = yval[i + 1];      // 寄存器中的值是可以循环利用的! 向左移一位!
        }

        // advancd on global idx
        idx  += nx;                     // 全局索引idx提步到下一个y轴位置!!!!
        __syncthreads();                // 同步线程块中的所有线程!!!
    }
}

// 多GPU实现2D波动方程u = u(x,y,t)的仿真计算.
int main(int argc, char** argv) {

    int ngpus;                                              // 多GPUs计算那么第一件事情就是确定设备中有多少个GPU!
    CUDA_ERROR_CHECK( hipGetDeviceCount(&ngpus) );
    printf("> CUDA-capable device count: %i\n", ngpus);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //  get it from command line
    if (argc > 1) {
        if (atoi(argv[1]) > ngpus) {    // 输入的要使用的GPU设备数量不能大于计算机所配备的GPU实际数量.
            fprintf(stderr, "Invalid number of GPUs specified: %d is greater than the total number of GPUs in this platform (%d)\n", atoi(argv[1]), ngpus);
            exit(1);
        }
        ngpus  = atoi(argv[1]);         // 计算得出真正用于2D波动方程仿真计算的GPU数量.
    }
    
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // size
    const int nsteps  = 3600;                                            // 定义有限差分迭代时间步数.
    const int nx      = 1024 * ngpus;                                    // 定义波场wave field u(x,y; t)在x维度的网格大小x.
    const int ny      = 1024 * ngpus;                                    // 定义波场wave field u(x,y; t)在y维度的网格大小y.
    const int iny     = ny / ngpus + PAD * 2;                            // 简化问题, 统一以y轴划分每个分区, 在边界左右两边都有NPAD的padding点. iny表示的是每个GPU分配的分区的y维度网格大小, 包括padding区域.

    size_t isize = nx * iny;                                            // 每个分区需要的网格点总数量.
    size_t ibyte = isize * sizeof(float);                               // 每个分区需要的设备内存字节数.
    size_t iexchange = PAD * nx * sizeof(float);                        // 需要在分区间进行数据交换的切片区域占据的内存字节数.
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // set up gpu card
    float *d_u2[ngpus], *d_u1[ngpus];                                   // 浮点指针数组d_u2和d_u1. 使用2个设备数组d_u1, d_u2. 一个数组用于保存当前波场wave field的状态, 另一个数组用于保存更新后的波场wave field的状态.
    for(int i = 0; i < ngpus; i++) {
        // set device
        CUDA_ERROR_CHECK( hipSetDevice(i) );                          // 不要忘记第一件事情就是set我们想要使用的GPU为当前GPU!!!  

        // allocate device memories   // 对每个GPU分配好需要的设备内存.
        CUDA_ERROR_CHECK( hipMalloc((void**)&d_u1[i], ibyte) );       // 每个GPU分配的分区的字节总数ibyte.
        CUDA_ERROR_CHECK( hipMalloc((void**)&d_u2[i], ibyte) );

        CUDA_ERROR_CHECK( hipMemset(d_u1[i], 0, ibyte) );             // 初始化波场状态值为0.
        CUDA_ERROR_CHECK( hipMemset(d_u2[i], 0, ibyte) );             // 初始化波场状态值为0.

        printf("GPU %i: allocated %.2f MB gmem\n", i, (4.f * ibyte) / (1024.f * 1024.f) );
        setup_coef ();                                                  // 配置有限差分系数.
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // stream definition
    hipStream_t stream_halo[ngpus], stream_body[ngpus];                // 只有halo区域的数据需要在GPU间进行相互传输. 因此我们创建多个独立的stream.
    // stream_halo数组用于halo区域的波场状态计算以及GPU间的数据传输; stream_body用于网格内点波场状态计算`.
    for (int i = 0; i < ngpus; i++) {
        CUDA_ERROR_CHECK( hipSetDevice(i) );                          // 不要忘记第一件事情就是set我们想要使用的GPU为当前GPU!!!  
        CUDA_ERROR_CHECK( hipStreamCreate(&stream_halo[i]) );
        CUDA_ERROR_CHECK( hipStreamCreate(&stream_body[i]) );
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // calculate index for computation. 计算halo区域和网格内点区域的区间索引!!!
    int haloStart[ngpus], bodyStart[ngpus], haloEnd[ngpus], bodyEnd[ngpus];
    calcIndex(haloStart, haloEnd, bodyStart, bodyEnd, ngpus, iny);      // iny表示的是每个GPU分配的分区的y维度网格大小, 包括padding区域.

    int src_skip[ngpus], dst_skip[ngpus];

    if(ngpus > 1) 
        calcSkips(src_skip, dst_skip, nx, iny);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // kernel launch configuration
    dim3 block(BDIMX);                          // 对于2D波动方程仿真, 这里采用的是1D thread block + 1D thread grid. 为什么不采用2D thread block和2D thread grid?
    dim3 grid(nx / block.x);                    // 1D线程网格的大小即 nx除以线程块的大小. 在x轴方向上并没有划分区域, 而是按照y轴方向划分区域.

    // set up event for timing                  // 记录时间.
    CUDA_ERROR_CHECK( hipSetDevice(0) );
    hipEvent_t start, stop;
    CUDA_ERROR_CHECK( hipEventCreate(&start) );
    CUDA_ERROR_CHECK( hipEventCreate(&stop ) );
    CUDA_ERROR_CHECK( hipEventRecord(start, 0) );

    // main loop for wave propagation
    for(int istep = 0; istep < nsteps; istep++) {     // 开始迭代! 以进行波动仿真!
        // save snap image
        if(istep == iMovie) 
            saveSnapshotIstep(istep, nx, ny, ngpus, d_u2);

        // add wavelet only onto gpu0
        if (istep == 0) {         // 在step = 0时, 引入initial value of wave.
            CUDA_ERROR_CHECK( hipSetDevice(0) );
            kernel_add_wavelet<<<grid, block>>>(d_u2[0], 20.0, nx, iny, ngpus);
        }

        // halo part. 波动仿真计算.
        for (int i = 0; i < ngpus; i++) {
            CUDA_ERROR_CHECK( hipSetDevice(i) );
            // compute halo
            kernel_2dfd<<<grid, block, 0, stream_halo[i]>>>(d_u1[i], d_u2[i], nx, haloStart[i], haloEnd[i]);
            // compute internal
            kernel_2dfd<<<grid, block, 0, stream_body[i]>>>(d_u1[i], d_u2[i], nx, bodyStart[i], bodyEnd[i]);
        }

        // exchange halo. halo区域数据交换..
        if (ngpus > 1) {
            CUDA_ERROR_CHECK( hipMemcpyAsync(d_u1[1] + dst_skip[0], d_u1[0] + src_skip[0], iexchange, hipMemcpyDefault, stream_halo[0]) );
            CUDA_ERROR_CHECK( hipMemcpyAsync(d_u1[0] + dst_skip[1], d_u1[1] + src_skip[1], iexchange, hipMemcpyDefault, stream_halo[1]) );
        }

        for (int i = 0; i < ngpus; i++) {         // 每一次迭代最后都要记得同步所有设备.
            CUDA_ERROR_CHECK( hipSetDevice(i) );
            CUDA_ERROR_CHECK( hipDeviceSynchronize() );

            float *tmpu0 = d_u1[i];               // 双缓冲策略, 一个数组用于保存当前波场wave field的状态, 另一个数组用于保存更新后的波场wave field的状态.
            d_u1[i] = d_u2[i];
            d_u2[i] = tmpu0;
        }
    }

    CUDA_ERROR_CHECK( hipDeviceSynchronize() );
    CUDA_ERROR_CHECK( hipGetLastError() );
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // clear
    for (int i = 0; i < ngpus; i++) {
        CUDA_ERROR_CHECK( hipSetDevice(i) );

        CUDA_ERROR_CHECK( hipStreamDestroy(stream_halo[i]) );
        CUDA_ERROR_CHECK( hipStreamDestroy(stream_body[i]) );

        CUDA_ERROR_CHECK( hipFree(d_u1[i]) );
        CUDA_ERROR_CHECK( hipFree(d_u2[i]) );

        CUDA_ERROR_CHECK( hipDeviceReset() );
    }

    return 0;
}
